#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void prefix_sum(volatile int *ds, int *da, int sz) {
  if (threadIdx.x >= sz - 1)
    return;
  int temp_val = 0;
  while (temp_val == 0) {
    temp_val = ds[threadIdx.x];
  }
  ds[threadIdx.x + 1] = ds[threadIdx.x] + da[threadIdx.x];
}

int main() {
  int a[] = {1, 2, 3, 4, 5, 6, 7, 8};
  int sz = 8;
  int *d_a;
  hipMalloc(&d_a, sz * sizeof(int));
  hipMemcpy(d_a, a, sz * sizeof(int), hipMemcpyHostToDevice);
  int *d_s;
  hipMalloc(&d_s, sz * sizeof(int));
  hipMemset(d_s, 0, sz * sizeof(int));
  hipMemcpy(d_s, a, sizeof(int), hipMemcpyHostToDevice);
  int s[sz];
  prefix_sum<<<1, 256>>>(d_s, d_a, sz);
  hipMemcpy(s, d_s, sz * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < sz; i++)
    cout << s[i] << ",";
  cout << endl;
  return 0;
}