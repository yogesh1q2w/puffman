#include "hip/hip_runtime.h"
#define MAX_THREADS 1024
#define MAX_FILE_NAME_SIZE 100
#define MAX_THREADS_TO_USE 4096

#include <iostream>
#include <fstream>
#include <math.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#include "huffman.h"
#include "decompressKernel.h"

using namespace std;

typedef unsigned long long int ull;

inline unsigned findNoOfThreadBlocks(unsigned totalNoOfThreads) {
	unsigned noOfThreadBlocks = ceil((double)totalNoOfThreads/MAX_THREADS);
	return noOfThreadBlocks;
}

unsigned char* calculateOffsetAndWriteOutput(unsigned char* input, unsigned size,
	unsigned blockSize, const HuffmanTree& tree, unsigned& outputSize) 
{
	HuffmanTree* treePtr;
	hipMalloc(&treePtr, sizeof(HuffmanTree));
	hipMemcpy(treePtr, &tree, sizeof(HuffmanTree), hipMemcpyHostToDevice);

	unsigned *offsets;
	unsigned char *dOutput, *output, *dInput, *dInputInBytes;
	hipMalloc(&dInput, size*sizeof(unsigned char));
	hipMemcpy(dInput, input, size*sizeof(unsigned char), hipMemcpyHostToDevice);
	
	hipMalloc(&dInputInBytes, size*8*sizeof(unsigned char));
	unsigned noOfThreads = ceil(((double)size)/blockSize);
	unsigned noOfThreadBlocks = findNoOfThreadBlocks(noOfThreads);
	
	convertBitsToBytes<<<noOfThreadBlocks, MAX_THREADS>>>(dInput, dInputInBytes, size,
		blockSize);
	hipDeviceSynchronize();
	hipFree(dInput);
	hipMalloc(&offsets, (noOfThreads+1)*sizeof(unsigned));
	
	calculateNoOfTokensInBlock<<<noOfThreadBlocks, MAX_THREADS>>>(dInputInBytes, size*8,
		blockSize*8, treePtr, offsets);
	hipDeviceSynchronize();
	// printOffsets<<<1,1>>>(offsets, noOfThreads+1);
	
	thrust::exclusive_scan(thrust::device, offsets, offsets+noOfThreads+1, offsets);
	// printOffsets<<<1,1>>>(offsets, noOfThreads+1);
	hipDeviceSynchronize();
	
	hipMemcpy(&outputSize, offsets+noOfThreads, sizeof(unsigned), hipMemcpyDeviceToHost);
	hipMalloc(&dOutput, outputSize*sizeof(unsigned char));
	output = new unsigned char[outputSize*sizeof(unsigned char)];
	writeOutput<<<noOfThreadBlocks, MAX_THREADS>>>(dInputInBytes, dOutput, size*8, blockSize*8,
		treePtr, offsets);
	hipDeviceSynchronize();
	
	hipFree(offsets);
	hipFree(dInputInBytes);

	hipMemcpy(output, dOutput, outputSize*sizeof(unsigned char), hipMemcpyDeviceToHost);
	// printf("--->%s\n",output);
	hipFree(dOutput);
	return output;
}

void readContentFromFile(ifstream& inputFile, ofstream& outputFile, const HuffmanTree& tree,
	unsigned blockSize, ull sizeOfInputFile) 
{
	size_t memoryFree,memoryTotal;
    hipError_t error;

    error = hipMemGetInfo(&memoryFree,&memoryTotal);
    if(error != hipSuccess) {
        printf("Error encountered: %s\n", hipGetErrorString(error));
        return;
    }

	const unsigned chunkSize = MAX_THREADS_TO_USE * blockSize;
	unsigned char input[chunkSize], *output;
	unsigned outputSize;
	ull sizeWrittenToFile = 0;

	while(inputFile) {
		inputFile.read((char *)input, chunkSize);
		unsigned noOfBytesRead = inputFile.gcount();
		output = calculateOffsetAndWriteOutput(input, noOfBytesRead, blockSize,
			tree, outputSize);
		ull sizeToWrite = min((ull)outputSize, sizeOfInputFile-sizeWrittenToFile);
		outputFile.write((char *)output, sizeToWrite);
		sizeWrittenToFile += sizeToWrite;
		delete[] output;
	}
}

int main(int argc, char* argv[]) {
	if(argc != 2) {
		cout << "The usage is ./a.out <fileToDecompress>" << endl;
		return 0;
	}
	char* filename = argv[1];
	ifstream file(filename, ios::in|ios::binary);
	if(!file) {
		cout << "The file could not be opened" << endl;
		return 0;
	}
	ull sizeOfFile;
	unsigned int blockSize;
	file.read((char *)&sizeOfFile, sizeof(ull));
	file.read((char *)&blockSize, sizeof(unsigned int));
	blockSize /= 8;
	// cout << sizeOfFile << ' ' << blockSize << endl;

	HuffmanTree tree;
	tree.readFromFile(file);
	ofstream outputFile("decompressed_output", ios::out|ios::binary);

	readContentFromFile(file, outputFile, tree, blockSize, sizeOfFile);
	file.close();
	outputFile.close();

	return 0;
}