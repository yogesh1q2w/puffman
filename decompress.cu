#include "hip/hip_runtime.h"
#include <chrono>
#include <fstream>
#include <iostream>
#include <math.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include "decompressKernel.h"
#include "huffman.h"

using namespace std;


inline unsigned findNoOfThreadBlocks(unsigned totalNoOfThreads) {
  unsigned noOfThreadBlocks = ceil((double)totalNoOfThreads / MAX_THREADS);
  return noOfThreadBlocks;
}

unsigned char *calculateOffsetAndWriteOutput(unsigned char *input,
                                             ull size, unsigned blockSize) {
  unsigned *offsets;
  unsigned char *dOutput, *dInput, *dInputInBytes;
  hipMalloc(&dInput, size);
  hipMemcpy(dInput, input, size, hipMemcpyHostToDevice);
  hipHostFree(input);
  hipMalloc(&dInputInBytes, size * 8);
  unsigned noOfThreads = ceil(((double)size) / blockSize);
  unsigned noOfThreadBlocks = findNoOfThreadBlocks(noOfThreads);

  convertBitsToBytes<<<noOfThreadBlocks, MAX_THREADS>>>(dInput, dInputInBytes,
                                                        size, blockSize);
  hipDeviceSynchronize();
  hipFree(dInput);
  hipMalloc(&offsets, (noOfThreads + 1) * sizeof(unsigned));

  calculateNoOfTokensInBlock<<<noOfThreadBlocks, MAX_THREADS>>>(
      dInputInBytes, size * 8, blockSize * 8, offsets);
  hipDeviceSynchronize();

  thrust::exclusive_scan(thrust::device, offsets, offsets + noOfThreads + 1,
                         offsets);
  hipDeviceSynchronize();

  uint outputSize;
  hipMemcpy(&outputSize, offsets + noOfThreads, sizeof(unsigned),
             hipMemcpyDeviceToHost);
  hipMalloc(&dOutput, outputSize);
  writeOutput<<<noOfThreadBlocks, MAX_THREADS>>>(
      dInputInBytes, dOutput, size * 8, blockSize * 8, offsets);
  hipDeviceSynchronize();

  hipFree(offsets);
  hipFree(dInputInBytes);

  unsigned char* output;
  hipHostMalloc(&output, outputSize);
  hipMemcpy(output, dOutput, outputSize,
             hipMemcpyDeviceToHost);
  hipFree(dOutput);
  return output;
}

ull findSizeOfInputFile(ifstream& inputFile) {
  streampos currentPositionInFile = inputFile.tellg();
  inputFile.seekg(0, inputFile.end);
  ull maxSizeOfInputFile = inputFile.tellg();
  inputFile.seekg(currentPositionInFile);
  return maxSizeOfInputFile;
}

void readContentFromFile(ifstream &inputFile, ofstream &outputFile,
                         const HuffmanTree &tree, unsigned blockSize,
                         ull sizeOfOriginalFile) {

  unsigned size = tree.treeInArray.size();
  hipMemcpyToSymbol(HIP_SYMBOL(deviceTree), tree.treeInArray.data(),
                     size * sizeof(TreeArrayNode));
  size--;
  hipMemcpyToSymbol(HIP_SYMBOL(rootIndex), &size, sizeof(int));

  ull maxSizeOfInputFile = ceil(findSizeOfInputFile(inputFile)/4.);
  cout << "MAXSIZEOFINPUTFILE: " << maxSizeOfInputFile << endl;
  unsigned int *input, *output;
  hipHostMalloc(&input, maxSizeOfInputFile);

  inputFile.read((char *)input, maxSizeOfInputFile);

  ull noOfBytesRead = inputFile.gcount();
  for(uint i=0; i<noOfBytesRead; i++) {
		for(uint j=0; j<32; j++)
			cout << ( 1 & (((uint)input[i]) >> (31-j)));
	}
  cout << "noOfBytesRead: " << noOfBytesRead << endl;
  output = calculateOffsetAndWriteOutput(input, noOfBytesRead, blockSize);

  outputFile.write((char *)output, sizeOfOriginalFile);
  hipHostFree(output);
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    cout << "The usage is ./a.out <fileToDecompress>" << endl;
    return 0;
  }
  char *filename = argv[1];
  ifstream file(filename, ios::in | ios::binary);
  if (!file) {
    cout << "The file could not be opened" << endl;
    return 0;
  }
  ull sizeOfFile;
  unsigned int blockSize;
  file.read((char *)&sizeOfFile, sizeof(ull));
  file.read((char *)&blockSize, sizeof(unsigned int));
  blockSize /= 8;
  cout << sizeOfFile << "," << blockSize << endl;

  HuffmanTree tree;
  tree.readFromFile(file);
  ofstream outputFile("decompressed_output", ios::out | ios::binary);

  readContentFromFile(file, outputFile, tree, blockSize, sizeOfFile);
  file.close();
  outputFile.close();

  return 0;
}