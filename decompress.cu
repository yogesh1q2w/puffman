#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/scan.h>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#include "decompressKernel.h"
#include "huffman.h"

using namespace std;



// inline unsigned findNoOfThreadBlocks(unsigned totalNoOfThreads) {
//   unsigned noOfThreadBlocks = ceil((double)totalNoOfThreads / MAX_THREADS);
//   return noOfThreadBlocks;
// }

// unsigned char *calculateOffsetAndWriteOutput(unsigned char *input,
//                                              ull size, unsigned blockSize) {
//   unsigned *offsets;
//   unsigned char *dOutput, *dInput, *dInputInBytes;
//   hipMalloc(&dInput, size);
//   hipMemcpy(dInput, input, size, hipMemcpyHostToDevice);
//   hipHostFree(input);
//   hipMalloc(&dInputInBytes, size * 8);
//   unsigned noOfThreads = ceil(((double)size) / blockSize);
//   unsigned noOfThreadBlocks = findNoOfThreadBlocks(noOfThreads);

//   convertBitsToBytes<<<noOfThreadBlocks, MAX_THREADS>>>(dInput, dInputInBytes,
//                                                         size, blockSize);
//   hipDeviceSynchronize();
//   hipFree(dInput);
//   hipMalloc(&offsets, (noOfThreads + 1) * sizeof(unsigned));

//   calculateNoOfTokensInBlock<<<noOfThreadBlocks, MAX_THREADS>>>(
//       dInputInBytes, size * 8, blockSize * 8, offsets);
//   hipDeviceSynchronize();

//   thrust::exclusive_scan(thrust::device, offsets, offsets + noOfThreads + 1,
//                          offsets);
//   hipDeviceSynchronize();

//   ull outputSize;
//   hipMemcpy(&outputSize, offsets + noOfThreads, sizeof(unsigned),
//              hipMemcpyDeviceToHost);
//   hipMalloc(&dOutput, outputSize);
//   writeOutput<<<noOfThreadBlocks, MAX_THREADS>>>(
//       dInputInBytes, dOutput, size * 8, blockSize * 8, offsets);
//   hipDeviceSynchronize();

//   hipFree(offsets);
//   hipFree(dInputInBytes);

//   unsigned char* output;
//   hipHostMalloc(&output, outputSize);
//   hipMemcpy(output, dOutput, outputSize,
//              hipMemcpyDeviceToHost);
//   hipFree(dOutput);
//   return output;
// }

// ull findSizeOfInputFile(ifstream& inputFile) {
//   streampos currentPositionInFile = inputFile.tellg();
//   inputFile.seekg(0, inputFile.end);
//   ull maxSizeOfInputFile = inputFile.tellg();
//   inputFile.seekg(currentPositionInFile);
//   return maxSizeOfInputFile;
// }

// void readContentFromFile(ifstream &inputFile, ofstream &outputFile,
//                          const HuffmanTree &tree, unsigned blockSize,
//                          ull sizeOfOriginalFile) {

//   unsigned size = tree.treeInArray.size();
//   hipMemcpyToSymbol(HIP_SYMBOL(deviceTree), tree.treeInArray.data(),
//                      size * sizeof(TreeArrayNode));
//   size--;
//   hipMemcpyToSymbol(HIP_SYMBOL(rootIndex), &size, sizeof(int));

//   ull maxSizeOfInputFile = findSizeOfInputFile(inputFile);
//   unsigned char *input, *output;
//   hipHostMalloc(&input, maxSizeOfInputFile);

//   inputFile.read((char *)input, maxSizeOfInputFile);
//   ull noOfBytesRead = inputFile.gcount();
//   output = calculateOffsetAndWriteOutput(input, noOfBytesRead, blockSize);

//   outputFile.write((char *)output, sizeOfOriginalFile);
//   hipHostFree(output);
// }

int main(int argc, char *argv[]) {
  if (argc != 2) {
    cout << "The usage is ./a.out <fileToDecompress>" << endl;
    return 0;
  }
  char *filename = argv[1];
  FILE *inputFile, *outputFile;
  inputFile = fopen(filename, "rb");
  if (!inputFile) {
    cout << "The file could not be opened" << endl;
    return 0;
  }
  outputFile = fopen("decompressed_output", "wb");

  ull sizeOfFile;
  unsigned int blockSize;
  if(1 != fread(&sizeOfFile, sizeof(ull), 1, inputFile))fatal("File read error 1");
  if(1 != fread(&blockSize, sizeof(uint), 1, inputFile))fatal("File read error 2");
  cout << sizeOfFile << "," << blockSize << endl;

  HuffmanTree tree;
  tree.readFromFile(inputFile);

  // readContentFromFile(file, outputFile, tree, blockSize, sizeOfFile);
  fclose(inputFile);
  fclose(outputFile);

  return 0;
}