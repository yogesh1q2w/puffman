#include "hip/hip_runtime.h"
#define MAX_THREADS 1024
#define MAX_FILE_NAME_SIZE 100
#define MAX_THREADS_TO_USE 65536

#include <chrono>
#include <fstream>
#include <iostream>
#include <math.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include "decompressKernel.h"
#include "huffman.h"

using namespace std;

typedef unsigned long long int ull;

inline unsigned findNoOfThreadBlocks(unsigned totalNoOfThreads) {
  unsigned noOfThreadBlocks = ceil((double)totalNoOfThreads / MAX_THREADS);
  return noOfThreadBlocks;
}

unsigned char *calculateOffsetAndWriteOutput(unsigned char *input,
                                             ull size, unsigned blockSize) {
  unsigned *offsets;
  unsigned char *dOutput, *dInput, *dInputInBytes;
  hipMalloc(&dInput, size);
  hipMemcpy(dInput, input, size, hipMemcpyHostToDevice);
  hipHostFree(input);
  hipMalloc(&dInputInBytes, size * 8);
  unsigned noOfThreads = ceil(((double)size) / blockSize);
  unsigned noOfThreadBlocks = findNoOfThreadBlocks(noOfThreads);

  convertBitsToBytes<<<noOfThreadBlocks, MAX_THREADS>>>(dInput, dInputInBytes,
                                                        size, blockSize);
  hipDeviceSynchronize();
  hipFree(dInput);
  hipMalloc(&offsets, (noOfThreads + 1) * sizeof(unsigned));

  calculateNoOfTokensInBlock<<<noOfThreadBlocks, MAX_THREADS>>>(
      dInputInBytes, size * 8, blockSize * 8, offsets);
  hipDeviceSynchronize();

  thrust::exclusive_scan(thrust::device, offsets, offsets + noOfThreads + 1,
                         offsets);
  hipDeviceSynchronize();

  ull outputSize;
  hipMemcpy(&outputSize, offsets + noOfThreads, sizeof(unsigned),
             hipMemcpyDeviceToHost);
  hipMalloc(&dOutput, outputSize);
  writeOutput<<<noOfThreadBlocks, MAX_THREADS>>>(
      dInputInBytes, dOutput, size * 8, blockSize * 8, offsets);
  hipDeviceSynchronize();

  hipFree(offsets);
  hipFree(dInputInBytes);

  unsigned char* output;
  hipHostMalloc(&output, outputSize);
  hipMemcpy(output, dOutput, outputSize,
             hipMemcpyDeviceToHost);
  hipFree(dOutput);
  return output;
}

void findSizeOfInputFile(ifstream& inputFile) {
  streampos currentPositionInFile = inputFile.tellg();
  inputFile.seekg(0, inputFile.end);
  ull maxSizeOfInputFile = inputFile.tellg();
  inputFile.seekg(currentPositionInFile);
  return maxSizeOfInputFile;
}

void readContentFromFile(ifstream &inputFile, ofstream &outputFile,
                         const HuffmanTree &tree, unsigned blockSize,
                         ull sizeOfOriginalFile) {

  unsigned size = tree.treeInArray.size();
  hipMemcpyToSymbol(HIP_SYMBOL(deviceTree), tree.treeInArray.data(),
                     size * sizeof(TreeArrayNode));
  size--;
  hipMemcpyToSymbol(HIP_SYMBOL(rootIndex), &size, sizeof(int));

  ull maxSizeOfInputFile = findSizeOfInputFile(inputFile);
  unsigned char *input, *output;
  hipHostMalloc(&input, maxSizeOfInputFile);

  inputFile.read((char *)input, maxSizeOfInputFile);
  ull noOfBytesRead = inputFile.gcount();
  output = calculateOffsetAndWriteOutput(input, noOfBytesRead, blockSize);

  outputFile.write((char *)output, sizeOfOriginalFile);
  hipHostFree(output);
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    cout << "The usage is ./a.out <fileToDecompress>" << endl;
    return 0;
  }
  char *filename = argv[1];
  ifstream file(filename, ios::in | ios::binary);
  if (!file) {
    cout << "The file could not be opened" << endl;
    return 0;
  }
  ull sizeOfFile;
  unsigned int blockSize;
  file.read((char *)&sizeOfFile, sizeof(ull));
  file.read((char *)&blockSize, sizeof(unsigned int));
  blockSize /= 8;
  cout << sizeOfFile << "," << blockSize << endl;

  HuffmanTree tree;
  tree.readFromFile(file);
  ofstream outputFile("decompressed_output", ios::out | ios::binary);

  readContentFromFile(file, outputFile, tree, blockSize, sizeOfFile);
  file.close();
  outputFile.close();

  return 0;
}