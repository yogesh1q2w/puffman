#define COMPRESS_CU
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#include "compressKernel.h"
#include "constants.h"
#include "huffman.h"

using namespace std;

uint blockSize = BLOCK_SIZE;
uint *fileContent, *dfileContent;
codedict *dictionary;
uint dictionarySize;
unsigned char useSharedMemory;
unsigned long long int fileSize; 
uint intFileSize;

void printDictionary(uint *frequency) {
  for (unsigned short i = 0; i < 256; i++) {
    if (frequency[i]) {
      cout << char(i) << "\t|\t" << frequency[i] << "\t|\t";
      for (unsigned char j = 0; j < dictionary->codeSize[i]; j++)
        cout << int(dictionary->code[j * 256 + i]) << ",";
      cout << endl;
    }
  }
}

void getFrequencies(uint *frequency) {

  uint *dfrequency;
  hipMalloc((void **)&dfrequency, 256 * sizeof(uint));
  CUERROR
  hipMemset(dfrequency, 0, 256 * sizeof(uint));
  CUERROR
  uint *d_PartialHistograms;
  hipMalloc((void **)&d_PartialHistograms,
             sizeof(uint) * HIST_BLOCK * HIST_SIZE);
  CUERROR
  TIMER_START(hist)
  cu_histgram<<<HIST_BLOCK, HIST_THREADS>>>(d_PartialHistograms, dfileContent,
                                            intFileSize, fileSize);
  mergeHistogram<<<HIST_BLOCK, HIST_SIZE>>>(dfrequency, d_PartialHistograms);
  hipMemcpy(frequency, dfrequency, 256 * sizeof(uint), hipMemcpyDeviceToHost);
  TIMER_STOP(hist)
  hipFree(d_PartialHistograms);
  CUERROR
  hipFree(dfrequency);
  CUERROR
}

void deepCopyHostToConstant() {
  hipMemcpyToSymbol(HIP_SYMBOL(const_code), dictionary->code,
                     dictionary->maxCodeSize * 256);
  CUERROR

  hipMemcpyToSymbol(HIP_SYMBOL(const_codeSize), dictionary->codeSize, 256);
  CUERROR
}

inline unsigned char getcharAt(uint pos) {
  return (fileContent[pos >> 2] >> ((pos & 3U) << 3)) & 0xFFU;
}

void getOffsetArray(uint *bitOffsets, uint &encodedFileSize) {
  bitOffsets[0] = 0;
  uint searchValue = BLOCK_SIZE;
  uint i;
  for (i = 1; i < fileSize; i++) {
    bitOffsets[i] = bitOffsets[i - 1] + dictionary->codeSize[getcharAt(i - 1)];

    if (bitOffsets[i] > searchValue) {
      bitOffsets[i - 1] = searchValue;
      searchValue += BLOCK_SIZE;
      i--;
    } else if (bitOffsets[i] == searchValue) {
      searchValue += BLOCK_SIZE;
    }
  }

  if (bitOffsets[i - 1] + dictionary->codeSize[getcharAt(i - 1)] >
      searchValue) {
    bitOffsets[i - 1] = searchValue;
    searchValue += BLOCK_SIZE;
  }
  encodedFileSize =
      bitOffsets[fileSize - 1] + dictionary->codeSize[getcharAt(fileSize - 1)];
}

void writeFileContents(FILE *outputFile) {

  uint *compressedFile, *d_compressedFile;
  uint *bitOffsets, *dbitOffsets;
  hipHostMalloc(&bitOffsets, fileSize * sizeof(uint));
  CUERROR
  hipMalloc((void **)&dbitOffsets, fileSize * sizeof(uint));
  CUERROR
  deepCopyHostToConstant();

  uint encodedFileSize;
  TIMER_START(offset)
  getOffsetArray(bitOffsets, encodedFileSize);
  TIMER_STOP(offset)

  hipMemcpy(dbitOffsets, bitOffsets, fileSize * sizeof(uint),
             hipMemcpyHostToDevice);
  CUERROR

  uint writeSize = (encodedFileSize + 31) >> 5;

  hipHostMalloc(&compressedFile, writeSize * sizeof(uint));
  CUERROR
  hipMalloc((void **)&d_compressedFile, writeSize * sizeof(uint));
  CUERROR

  if (useSharedMemory) {
    TIMER_START(kernel)
    skss_compress_with_shared<<<BLOCK_NUM, 256,
                                (dictionary->maxCodeSize + 1) * 256>>>(
        fileSize, dfileContent, dbitOffsets, d_compressedFile,
        dictionary->maxCodeSize);
    TIMER_STOP(kernel)
  } else {
    TIMER_START(kernel)
    skss_compress<<<BLOCK_NUM, 256>>>(fileSize, dfileContent, dbitOffsets,
                                      d_compressedFile,
                                      dictionary->maxCodeSize);
    TIMER_STOP(kernel)
  }

  hipMemcpy(compressedFile, d_compressedFile, writeSize * sizeof(uint),
             hipMemcpyDeviceToHost);
  CUERROR
  fwrite(compressedFile, sizeof(uint), writeSize, outputFile);
  fdatasync(outputFile->_fileno);
  hipFree(d_compressedFile);
  CUERROR
  hipHostFree(compressedFile);
  CUERROR
  hipFree(dbitOffsets);
  CUERROR
  hipHostFree(bitOffsets);
  CUERROR
}

void readFile(uint *&fileContent, uint *&dfileContent, FILE *inputFile) {
  fseek(inputFile, 0L, SEEK_END);
  fileSize = ftell(inputFile);
  fseek(inputFile, 0L, SEEK_SET);
  intFileSize = (fileSize + 3) >> 2;
  hipHostMalloc(&fileContent, sizeof(uint) * intFileSize);
  CUERROR
  hipMalloc((void **)&dfileContent, sizeof(uint) * intFileSize);
  CUERROR
  uint sizeRead =
      fread(fileContent, sizeof(unsigned char), fileSize, inputFile);
  if (sizeRead != fileSize) {
    cout << "Error in reading the file. Aborting..." << endl;
    exit(0);
  }
  hipMemcpy(dfileContent, fileContent, sizeof(uint) * intFileSize,
             hipMemcpyHostToDevice);
  CUERROR
}

int main(int argc, char **argv) {
  FILE *inputFile, *outputFile;
  if (argc != 2) {
    cout << "Running format is ./compress <file name>" << endl;
    return 0;
  }

  inputFile = fopen(argv[1], "rb");

  if (!inputFile) {
    cout << "Please give a valid file to open." << endl;
    return 0;
  }

  uint frequency[256];

  TIMER_START(readFile)
  readFile(fileContent, dfileContent, inputFile);
  TIMER_STOP(readFile)
  fsync(inputFile->_fileno);

  getFrequencies(frequency); // build histogram in GPU

  HuffmanTree tree;
  TIMER_START(tree)
  tree.HuffmanCodes(frequency, dictionary); // build Huffman tree in Host
  TIMER_STOP(tree)
  uint sharedMemoryPerBlock;
  hipDeviceGetAttribute((int *)&sharedMemoryPerBlock,
                         hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

  dictionarySize = dictionary->getSize();
  if (sharedMemoryPerBlock > dictionarySize) {
    useSharedMemory = 1;
  } else {
    useSharedMemory = 0;
  }

  cout << "Shared memory using bit is " << int(useSharedMemory) << endl;

  outputFile = fopen("compressed_output.bin", "wb");

  TIMER_START(meta)
  fwrite(&fileSize, sizeof(unsigned long long int), 1, outputFile);
  fwrite(&blockSize, sizeof(uint), 1, outputFile);
  fwrite(&tree.noOfLeaves, sizeof(uint), 1, outputFile);
  tree.writeTree(outputFile);
  TIMER_STOP(meta)
  writeFileContents(outputFile);
  hipHostFree(fileContent);
  CUERROR
  hipFree(dfileContent);
  CUERROR
  fclose(inputFile);
  fclose(outputFile);
  return 0;
}