#include "hip/hip_runtime.h"
#include "../include/compress_utils.cuh"
#include <vector>
using namespace std;
//---------------------------------HISTOGRAM-------------------------------------

inline __device__ void addByte(uint *s_WarpHist, unsigned char data) {
  atomicAdd(s_WarpHist + data, 1);
}

inline __device__ void addWord(uint *s_WarpHist, uint data) {
  addByte(s_WarpHist, (data >> 0) & 0xFFU);
  addByte(s_WarpHist, (data >> 8) & 0xFFU);
  addByte(s_WarpHist, (data >> 16) & 0xFFU);
  addByte(s_WarpHist, (data >> 24) & 0xFFU);
}

__global__ void cu_histgram(uint *d_PartialHistograms, uint *d_Data,
                            uint dataCount, uint byteCount) {

  __shared__ uint s_Hist[S_HIST_SIZE];
  uint *s_WarpHist = s_Hist + (threadIdx.x >> 5) * HIST_SIZE;
  uint warpLane = threadIdx.x & 31;

  for (uint i = warpLane; i < HIST_SIZE; i += WARP_SIZE) {
    s_WarpHist[i] = 0;
  }
  __syncthreads();

  uint pos = 0;
  for (pos = (blockIdx.x * blockDim.x) + threadIdx.x; pos < dataCount - 1;
       pos += (blockDim.x * gridDim.x)) {
    uint data = d_Data[pos];
    addWord(s_WarpHist, data);
  }

  if (pos == dataCount - 1) {
    uint data = d_Data[pos];
    switch (byteCount & 3) {
    case 1:
      addByte(s_WarpHist, (data >> 0) & 0xFFU);
      break;
    case 2:
      addByte(s_WarpHist, (data >> 0) & 0xFFU);
      addByte(s_WarpHist, (data >> 8) & 0xFFU);
      break;
    case 3:
      addByte(s_WarpHist, (data >> 0) & 0xFFU);
      addByte(s_WarpHist, (data >> 8) & 0xFFU);
      addByte(s_WarpHist, (data >> 16) & 0xFFU);
      break;
    default:
      addByte(s_WarpHist, (data >> 0) & 0xFFU);
      addByte(s_WarpHist, (data >> 8) & 0xFFU);
      addByte(s_WarpHist, (data >> 16) & 0xFFU);
      addByte(s_WarpHist, (data >> 24) & 0xFFU);
    }
  }

  __syncthreads();

  //
  for (uint bin = threadIdx.x; bin < HIST_SIZE; bin += HIST_THREADS) {
    uint sum = 0;
    for (uint i = 0; i < WARP_COUNT; i++) {
      sum += s_Hist[bin + i * HIST_SIZE];
    }
    d_PartialHistograms[blockIdx.x * HIST_SIZE + bin] = sum;
  }
}

__global__ void mergeHistogram(uint *d_Histogram, uint *d_PartialHistograms) {

  uint val = d_PartialHistograms[blockIdx.x * HIST_SIZE + threadIdx.x];
  atomicAdd(d_Histogram + threadIdx.x, val);
}

//-----------------------------------------------------------------------------------------------
__host__ inline unsigned char getcharAt(uint *dfileContent, uint pos) {
  return (dfileContent[pos >> 2] >> ((pos & 3U) << 3)) & 0xFFU;
}

__global__ void encode(uint fileSize, uint *dfileContent, uint *dblockCharPos,
                       uint *d_compressedFile, uint *d_dictionary_code,
                       unsigned char *d_dictionary_codelens, uint numBlocks) {
  uint block_idx;
  uint inputfile_idx;
  __shared__ struct codedict sh_dictionary;

  sh_dictionary.code[threadIdx.x] = d_dictionary_code[threadIdx.x];
  sh_dictionary.codeSize[threadIdx.x] = d_dictionary_codelens[threadIdx.x];

  __syncthreads();

  block_idx = (blockIdx.x * blockDim.x + threadIdx.x);

  while (block_idx < numBlocks) {
    inputfile_idx = dblockCharPos[block_idx];
    uint input = dfileContent[inputfile_idx / 4];
    uint bits_written = 0;
    uint changeIndex = (block_idx * BLOCK_SIZE) >> 5;
    uint window = 0;
    uint window_pos = 0;
    while (bits_written < BLOCK_SIZE && inputfile_idx <= fileSize) {
      uint code = sh_dictionary.code[GET_CHAR(input, inputfile_idx & 3)];
      unsigned char code_len =
          sh_dictionary.codeSize[GET_CHAR(input, inputfile_idx & 3)];
      inputfile_idx++;
      if ((inputfile_idx & 3) == 0 && inputfile_idx < fileSize)
        input = dfileContent[inputfile_idx / 4];
      while (window_pos + code_len < INT_BITS && inputfile_idx <= fileSize) {
        window <<= code_len;
        window |= code;
        window_pos += code_len;

        if (inputfile_idx < fileSize) {
          code = sh_dictionary.code[GET_CHAR(input, inputfile_idx & 3)];
          code_len = sh_dictionary.codeSize[GET_CHAR(input, inputfile_idx & 3)];
          inputfile_idx++;
          if ((inputfile_idx & 3) == 0 && inputfile_idx < fileSize)
            input = dfileContent[inputfile_idx / 4];
        }
      }
      const int diff = window_pos + code_len - INT_BITS;
      if (diff >= 0) {
        window <<= (code_len - diff);
        window |= (code >> diff);
        d_compressedFile[changeIndex++] |= window;
        window = code & ~(~0 << diff);
        window_pos = diff;
      } else {
        window <<= code_len;
        window |= code;
        const int shift = INT_BITS - (window_pos + code_len);
        window <<= shift;
        d_compressedFile[changeIndex++] |= window;
        window_pos = 0;
      }
      bits_written += 32;
    }
    block_idx += (BLOCK_NUM * NUM_THREADS);
  }
}

void readFile(uint *&fileContent, uint *&dfileContent, FILE *inputFile,
              unsigned long long int &fileSize, uint &intFileSize) {
  fseek(inputFile, 0L, SEEK_END);
  fileSize = ftell(inputFile);
  fseek(inputFile, 0L, SEEK_SET);
  intFileSize = (fileSize + 3) >> 2;
  hipHostMalloc(&fileContent, sizeof(uint) * intFileSize);
  CUERROR
  hipMalloc((void **)&dfileContent, sizeof(uint) * intFileSize);
  CUERROR
  if (fileSize !=
      fread(fileContent, sizeof(unsigned char), fileSize, inputFile))
    fatal("File read error 1");
  GPU_TIMER_START(HtD1)
  hipMemcpy(dfileContent, fileContent, sizeof(uint) * intFileSize,
             hipMemcpyHostToDevice);
  GPU_TIMER_STOP(HtD1)
  CUERROR
}

void getFrequencies(uint *dfileContent, unsigned long long int &fileSize,
                    uint *&frequency, uint &intFileSize) {
  hipHostMalloc(&frequency, 256 * sizeof(uint));
  uint *dfrequency;
  hipMalloc((void **)&dfrequency, 256 * sizeof(uint));
  hipMemset(dfrequency, 0, 256 * sizeof(uint));
  uint *d_PartialHistograms;
  hipMalloc((void **)&d_PartialHistograms,
             sizeof(uint) * HIST_BLOCK * HIST_SIZE);
  GPU_TIMER_START(hist)
  cu_histgram<<<HIST_BLOCK, HIST_THREADS>>>(d_PartialHistograms, dfileContent,
                                            intFileSize, fileSize);
  mergeHistogram<<<HIST_BLOCK, HIST_SIZE>>>(dfrequency, d_PartialHistograms);
  GPU_TIMER_STOP(hist)
  hipMemcpy(frequency, dfrequency, 256 * sizeof(uint), hipMemcpyDeviceToHost);
  hipFree(d_PartialHistograms);
  hipFree(dfrequency);
  CUERROR
}

void getOffsetArray(vector<unsigned int> &blockCharPos,
                    unsigned long long int &encodedFileSize,
                    unsigned long long int &fileSize, codedict &dictionary,
                    uint *fileContent) {
  blockCharPos.push_back(0);
  unsigned long long int searchValue = BLOCK_SIZE;
  unsigned long long int i;
  uint offset_sum = 0;
  for (i = 1; i <= fileSize; i++) {
    offset_sum += dictionary.codeSize[getcharAt(fileContent, i - 1)];

    if (offset_sum > searchValue) {
      blockCharPos.push_back(i - 1);
      offset_sum = searchValue;
      searchValue += BLOCK_SIZE;
      i--;
    } else if (offset_sum == searchValue) {
      blockCharPos.push_back(i);
      searchValue += BLOCK_SIZE;
    }
  }
  encodedFileSize = offset_sum;
}

void writeFileContents(FILE *outputFile, unsigned long long int &fileSize,
                       uint *fileContent, uint *dfileContent,
                       codedict &dictionary) {

  uint *compressedFile, *d_compressedFile;
  vector<unsigned int> blockCharPos;
  uint *dblockCharPos;

  unsigned long long int encodedFileSize;
  CPU_TIMER_START(offset)
  getOffsetArray(blockCharPos, encodedFileSize, fileSize, dictionary,
                 fileContent);
  CPU_TIMER_STOP(offset)

  uint numBlocks = blockCharPos.size();
  hipMalloc((void **)&dblockCharPos, numBlocks * sizeof(uint));
  hipMemcpy(dblockCharPos, &blockCharPos[0], numBlocks * sizeof(uint),
             hipMemcpyHostToDevice);
  CUERROR
  uint writeSize = (encodedFileSize + 31) >> 5;

  printf("Last offset = %d\n", blockCharPos[numBlocks-1]);

  hipHostMalloc(&compressedFile, writeSize * sizeof(uint));
  hipMalloc((void **)&d_compressedFile, writeSize * sizeof(uint));
  hipMemset(d_compressedFile, 0, writeSize * sizeof(uint));
  CUERROR

  uint *d_dictionary_code;
  unsigned char *d_dictionary_codelens;
  hipMalloc(&d_dictionary_code, 256 * sizeof(uint));
  hipMalloc(&d_dictionary_codelens, 256 * sizeof(unsigned char));
  hipMemcpy(d_dictionary_code, dictionary.code, 256 * sizeof(uint),
             hipMemcpyHostToDevice);
  hipMemcpy(d_dictionary_codelens, dictionary.codeSize,
             256 * sizeof(unsigned char), hipMemcpyHostToDevice);

  GPU_TIMER_START(kernel)
  encode<<<BLOCK_NUM, NUM_THREADS>>>(fileSize, dfileContent, dblockCharPos,
                                     d_compressedFile, d_dictionary_code,
                                     d_dictionary_codelens, numBlocks);
  GPU_TIMER_STOP(kernel)
  CUERROR
  hipMemcpy(compressedFile, d_compressedFile, writeSize * sizeof(uint),
             hipMemcpyDeviceToHost);
  CUERROR
  fwrite(&encodedFileSize, sizeof(unsigned long long int), 1, outputFile);
  fwrite(compressedFile, sizeof(uint), writeSize, outputFile);
  hipHostFree(compressedFile);
  hipHostFree(fileContent);
  hipFree(dblockCharPos);
  hipFree(d_compressedFile);
  hipFree(dfileContent);
  hipFree(d_dictionary_code);
  hipFree(d_dictionary_codelens);
  CUERROR
}
