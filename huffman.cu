#include "hip/hip_runtime.h"
#include "huffman.h"

#include <assert.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <queue>
#include <stdio.h>
#include <string.h>
#include <utility>

using namespace std;

void convertBitsToBytes(unsigned char *arrayInBits, unsigned char *arrayInBytes,
                        unsigned size) {
  unsigned sizeInBytes = ceil(size / 8.);
  for (unsigned i = 0; i < sizeInBytes; i++)
    for (unsigned j = 0; j < 8 && i * 8 + j < size; j++)
      arrayInBytes[i * 8 + j] = (arrayInBits[i] >> (7 - j)) & 1;
}

template <class T>
inline const T minHeapPop(priority_queue<T, vector<T>, greater<T>> &heap) {
  const T top = heap.top();
  heap.pop();
  return top;
}

unsigned char readByte(unsigned char *byte) {
  unsigned char result = 0;
  for (unsigned i = 0; i < 8; i++) {
    assert(byte[i] == 0 || byte[i] == 1);
    result = (result << 1) | byte[i];
  }
  return result;
}

TreeNode::TreeNode(unsigned char token, TreeNode *left = nullptr,
                   TreeNode *right = nullptr) {
  this->token = token;
  this->left = left;
  this->right = right;
}

void TreeArrayNode::assignValues(unsigned char token, int left = -1,
                                 int right = -1) {
  this->token = token;
  this->left = left;
  this->right = right;
}

HuffmanTree::HuffmanTree() {
  root = nullptr;
  noOfLeaves = 0;
}

HuffmanTree::~HuffmanTree() { deleteTree(root); }

void HuffmanTree::deleteTree(TreeNode *node) {
  if (node) {
    deleteTree(node->left);
    deleteTree(node->right);
    delete node;
  }
}

int HuffmanTree::createTreeFromFile(unsigned char *array, unsigned size,
                                    unsigned &offset, int &index) {
  unsigned char token = 0;
  assert(offset < size);
  assert(array[offset] == 0 || array[offset] == 1);
  if (array[offset] == 1) {
    token = readByte(&array[offset + 1]);
    offset += 9;
    treeInArray[index].assignValues(token);
  } else {
    offset++;
    int left = createTreeFromFile(array, size, offset, index);
    int right = createTreeFromFile(array, size, offset, index);
    treeInArray[index].assignValues(0, left, right);
  }
  return index++;
}

void HuffmanTree::readFromFile(ifstream &file) {
  unsigned int noOfLeaves;
  file.read((char *)&noOfLeaves, sizeof(unsigned int));
  unsigned sizeOfHuffman = 10 * noOfLeaves - 1;
  unsigned char huffmanTreeInBits[(unsigned)ceil(sizeOfHuffman / 8.)];
  file.read((char *)huffmanTreeInBits, (unsigned)ceil(sizeOfHuffman / 8.));

  unsigned char huffmanTreeInBytes[sizeOfHuffman];

  convertBitsToBytes(huffmanTreeInBits, huffmanTreeInBytes, sizeOfHuffman);

  treeInArray.resize(2 * noOfLeaves - 1);
  unsigned offset = 0;
  int index = 0;
  createTreeFromFile(huffmanTreeInBytes, sizeOfHuffman, offset, index);
}

void HuffmanTree::buildTreeFromFrequencies(unsigned long long int *frequency) {
  typedef pair<ull, TreeNode *> pullt;
  priority_queue<pullt, vector<pullt>, greater<pullt>> minHeap;
  noOfLeaves = 0;
  for (unsigned int i = 0; i < 256; i++) {
    if (frequency[i] > 0) {
      TreeNode *node = new TreeNode(i);
      minHeap.push(make_pair(frequency[i], node));
      noOfLeaves++;
    }
  }
  while (minHeap.size() > 1) {
    pullt pair1 = minHeapPop(minHeap);
    pullt pair2 = minHeapPop(minHeap);
    TreeNode *node = new TreeNode(0, pair1.second, pair2.second);
    ull newNodeFrequency = pair1.first + pair2.first;
    minHeap.push(make_pair(newNodeFrequency, node));
  }
  root = minHeap.top().second;
}

void HuffmanTree::getCodes(TreeNode *node, unsigned char *code,
                           unsigned char len, codedict *&dictionary) {
  if ((node->left == nullptr) && (node->right == nullptr)) {
    dictionary->codeSize[node->token] = len;
    memcpy(&dictionary->code[dictionary->maxCodeSize * node->token], code, len);
    dictionary->addCode(node->token, len, code);
    return;
  }

  if (node->left != nullptr) {
    code[len] = 0;
    getCodes(node->left, code, len + 1, dictionary);
  }

  if (node->right != nullptr) {
    code[len] = 1;
    getCodes(node->right, code, len + 1, dictionary);
  }
}

unsigned char HuffmanTree::_heightOfTree(TreeNode *node) {
  if (node == nullptr)
    return 0;
  unsigned char lHeight = _heightOfTree(node->left);
  unsigned char rHeight = _heightOfTree(node->right);
  return 1 + max(lHeight, rHeight);
}

unsigned char HuffmanTree::heightOfTree() { return _heightOfTree(root) - 1; }

void HuffmanTree::HuffmanCodes(unsigned long long int *freq,
                               codedict *&dictionary) {
  buildTreeFromFrequencies(freq);
  unsigned char maxCodeSize = heightOfTree();
  cout << "height of tree is " << int(maxCodeSize) << endl;
  unsigned char code[255];
  dictionary = new codedict(0, maxCodeSize);
  cout << "Object created" << endl;
  getCodes(root, code, 0, dictionary);
  cout << "Codes written" << endl;
}

void HuffmanTree::constructTree(TreeNode *node, unsigned char *bitsRepTree,
                                unsigned int *pos) {
  if ((node->left == nullptr) && (node->right == nullptr)) {
    bitsRepTree[(*pos)++] = 1;
    for (unsigned char i = 0; i < 8; i++)
      bitsRepTree[(*pos)++] = (node->token >> (7 - i)) & 1;
  } else {
    bitsRepTree[(*pos)++] = 0;
    constructTree(node->left, bitsRepTree, pos);
    constructTree(node->right, bitsRepTree, pos);
  }
}

void HuffmanTree::writeTree(ofstream &fptr) {
  unsigned char bitsRepTree[10 * noOfLeaves - 1];
  unsigned int pos = 0;
  constructTree(root, bitsRepTree, &pos);
  unsigned int writeTreeSize = ceil(pos / 8.);
  unsigned char finalTree[writeTreeSize];
  for (unsigned int i = 0; i < writeTreeSize; i++) {
    for (unsigned int j = 0; j < 8; j++) {
      if (bitsRepTree[i * 8 + j])
        finalTree[i] = (finalTree[i] << 1) | 1;
      else
        finalTree[i] = finalTree[i] << 1;
    }
  }
  fptr.write((char *)finalTree, writeTreeSize);
}

codedict::codedict(unsigned char _onDevice, unsigned char _maxCodeSize) {
  onDevice = _onDevice;
  maxCodeSize = _maxCodeSize;
  if (onDevice) {
    hipMalloc(&code, 256 * maxCodeSize * sizeof(unsigned char));
    hipError_t error = hipGetLastError();
    cout << "Error encountered: " << hipGetErrorString(error) << endl;
    hipMalloc(&codeSize, 256 * sizeof(unsigned char));
    error = hipGetLastError();
    cout << "Error encountered: " << hipGetErrorString(error) << endl;
  } else {
    code = new unsigned char[256 * maxCodeSize];
    codeSize = new unsigned char[256];
  }
}

void codedict::addCode(const unsigned char &token, const unsigned char &codeLen,
                       const unsigned char *sCode) {
  memcpy(&code[token * maxCodeSize], sCode, codeLen * sizeof(unsigned char));
}

void codedict::deepCopyHostToDevice(codedict *&destination) {
  hipMemcpy(destination->code, code, 256 * maxCodeSize * sizeof(unsigned char),
             hipMemcpyHostToDevice);
  hipError_t error = hipGetLastError();
  cout << "Error encountered: " << hipGetErrorString(error) << endl;
  hipMemcpy(destination->codeSize, codeSize, 256, hipMemcpyHostToDevice);
  error = hipGetLastError();
  cout << "Error encountered: " << hipGetErrorString(error) << endl;
}

unsigned short codedict::getSize() { return (256 * (maxCodeSize + 1) + 2); }

codedict::~codedict() {
  if (onDevice) {
    hipFree(code);
    hipFree(codeSize);
  } else {
    delete code;
    delete codeSize;
  }
}