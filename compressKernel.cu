#include "hip/hip_runtime.h"
#define KERNEL_CU
#define PER_THREAD_PROC 8
#define SEGMENT_SIZE 256
#include "compressKernel.h"

__constant__ unsigned char const_code[256 * 255];
__constant__ unsigned char const_codeSize[256];

__global__ void updatefrequency(unsigned int fileSize,
                                unsigned char *fileContent,
                                unsigned long long int *frequency) {
  unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int index = (id % SEGMENT_SIZE) +
                       (SEGMENT_SIZE * PER_THREAD_PROC * (id / SEGMENT_SIZE));
  for (unsigned int i = 0; i < PER_THREAD_PROC; i++) {
    if (index < fileSize) {
      atomicAdd(&frequency[fileContent[index]], 1);
      index += SEGMENT_SIZE;
    } else {
      break;
    }
  }
}

// __global__ void genBitCompressed(unsigned int lastBlockIndex,
//                                  unsigned char *dfileContent,
//                                  unsigned int *dbitOffsets,
//                                  unsigned char *dbitCompressedFile) {
//   unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
//   unsigned int index = (id % SEGMENT_SIZE) +
//                        (SEGMENT_SIZE * PER_THREAD_PROC * (id /
//                        SEGMENT_SIZE));
//   for (unsigned char i = 0; i < PER_THREAD_PROC; i++) {
//     if (index <= lastBlockIndex) {

//       if (index < lastBlockIndex) {
//         for (unsigned char j = 0; j < const_codeSize[dfileContent[index]];
//         j++)
//           dbitCompressedFile[dbitOffsets[index] + j] =
//               const_code[dfileContent[index]][j];
//       }

//       if (index > 0 &&
//           dbitOffsets[index - 1] + const_codeSize[dfileContent[index - 1]] !=
//               dbitOffsets[index]) {
//         unsigned int start =
//             dbitOffsets[index - 1] + const_codeSize[dfileContent[index - 1]];
//         for (unsigned int j = start; j < dbitOffsets[index]; j++)
//           dbitCompressedFile[j] = const_code[dfileContent[index]][j - start];
//       }
//       index += SEGMENT_SIZE;
//     } else {
//       break;
//     }
//   }
// }

// __global__ void encode(unsigned int bitCompressedFileSize,
//                        unsigned char *dbitCompressedFile,
//                        unsigned char *d_compressedFile) {
//   unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
//   unsigned int index = (id % SEGMENT_SIZE) +
//                        (SEGMENT_SIZE * PER_THREAD_PROC * (id /
//                        SEGMENT_SIZE));
//   for (unsigned int i = 0; i < PER_THREAD_PROC; i++) {
//     if (index < bitCompressedFileSize) {
//       for (unsigned int j = 0; j < 8; j++) {
//         if (dbitCompressedFile[index * 8 + j])
//           d_compressedFile[index] = (d_compressedFile[index] << 1) | 1;
//         else
//           d_compressedFile[index] = d_compressedFile[index] << 1;
//       }
//       index += SEGMENT_SIZE;
//     } else {
//       break;
//     }
//   }
// }

__global__ void skss_compress_with_shared(unsigned int lastBlockIndex,
                                          unsigned char *dfileContent,
                                          unsigned int *dbitOffsets,
                                          unsigned int *d_compressedFile,
                                          unsigned char maxCodeSize) {
  extern __shared__ unsigned char sh_dictionary[];
  memcpy(sh_dictionary, const_codeSize, 256);
  memcpy(&sh_dictionary[256], const_code, maxCodeSize * 256);
  unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int index = (id % SEGMENT_SIZE) +
                       (SEGMENT_SIZE * PER_THREAD_PROC * (id / SEGMENT_SIZE));
  for (unsigned int i = 0; i < PER_THREAD_PROC; i++) {
    if (index <= lastBlockIndex) {
      if (index < lastBlockIndex) {
        for (unsigned int j = 0; j < sh_dictionary[dfileContent[index]]; j++) {
          unsigned int compressedFilePos =
              (dbitOffsets[index] + j) / (8. * sizeof(unsigned int));
          unsigned int modifyIndex =
              ((dbitOffsets[index] + j) % (8 * sizeof(unsigned int)));
          modifyIndex = 8 * (modifyIndex / 8) + 7 - (modifyIndex % 8);
          unsigned int mask = 1 << modifyIndex;
          if (sh_dictionary[dfileContent[index] * maxCodeSize + j + 256]) {
            atomicOr(&d_compressedFile[compressedFilePos], mask);
          } else {
            atomicAnd(&d_compressedFile[compressedFilePos], ~mask);
          }
        }
      }

      if (index > 0 &&
          dbitOffsets[index - 1] + sh_dictionary[dfileContent[index - 1]] !=
              dbitOffsets[index]) {
        unsigned int start =
            dbitOffsets[index - 1] + sh_dictionary[dfileContent[index - 1]];
        for (unsigned int j = start; j < dbitOffsets[index]; j++) {
          unsigned int compressedFilePos = (j / (8. * sizeof(unsigned int)));
          unsigned int modifyIndex = (j % (8 * sizeof(unsigned int)));
          modifyIndex = 8 * (modifyIndex / 8) + 7 - (modifyIndex % 8);
          unsigned int mask = 1 << modifyIndex;
          if (sh_dictionary[dfileContent[index] * maxCodeSize + j - start +
                            256]) {
            atomicOr(&d_compressedFile[compressedFilePos], mask);
          } else {
            atomicAnd(&d_compressedFile[compressedFilePos], ~mask);
          }
        }
      }
      index += SEGMENT_SIZE;
    } else {
      break;
    }
  }
}

__global__ void skss_compress(unsigned int lastBlockIndex,
                              unsigned char *dfileContent,
                              unsigned int *dbitOffsets,
                              unsigned int *d_compressedFile,
                              unsigned char maxCodeSize) {
  unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int index = (id % SEGMENT_SIZE) +
                       (SEGMENT_SIZE * PER_THREAD_PROC * (id / SEGMENT_SIZE));
  for (unsigned int i = 0; i < PER_THREAD_PROC; i++) {
    if (index <= lastBlockIndex) {
      if (index < lastBlockIndex) {
        for (unsigned int j = 0; j < const_codeSize[dfileContent[index]]; j++) {
          unsigned int compressedFilePos =
              (dbitOffsets[index] + j) / (8. * sizeof(unsigned int));
          unsigned int modifyIndex =
              ((dbitOffsets[index] + j) % (8 * sizeof(unsigned int)));
          modifyIndex = 8 * (modifyIndex / 8) + 7 - (modifyIndex % 8);
          unsigned int mask = 1 << modifyIndex;
          if (const_code[dfileContent[index] * maxCodeSize + j]) {
            atomicOr(&d_compressedFile[compressedFilePos], mask);
          } else {
            atomicAnd(&d_compressedFile[compressedFilePos], ~mask);
          }
        }
      }

      if (index > 0 &&
          dbitOffsets[index - 1] + const_codeSize[dfileContent[index - 1]] !=
              dbitOffsets[index]) {
        unsigned int start =
            dbitOffsets[index - 1] + const_codeSize[dfileContent[index - 1]];
        for (unsigned int j = start; j < dbitOffsets[index]; j++) {
          unsigned int compressedFilePos = (j / (8. * sizeof(unsigned int)));
          unsigned int modifyIndex = (j % (8 * sizeof(unsigned int)));
          modifyIndex = 8 * (modifyIndex / 8) + 7 - (modifyIndex % 8);
          unsigned int mask = 1 << modifyIndex;
          if (const_code[dfileContent[index] * maxCodeSize + j - start]) {
            atomicOr(&d_compressedFile[compressedFilePos], mask);
          } else {
            atomicAnd(&d_compressedFile[compressedFilePos], ~mask);
          }
        }
      }
      index += SEGMENT_SIZE;
    } else {
      break;
    }
  }
}

__global__ void printDict(codedict &dict) {
  printf("MaxcodeSize = %u", dict.maxCodeSize);
  for (int i = 0; i < 256; i++) {
    if (dict.codeSize[i] > 0) {
      printf("%c-> ", i);
      for (int j = 0; j < dict.codeSize[i]; j++)
        printf("%u", dict.code[i * dict.maxCodeSize + j]);
      printf("\n");
    }
  }
}